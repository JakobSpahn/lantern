#include "lantern/tensor/accel/cuda/CUDATensor.h"

#include "lantern/tensor/accel/cuda/CUDABackend.h"

#include <string>
#include <cassert>
#include <memory>

namespace lt {

CUDATensor::~CUDATensor() {
    hipFree(arr_);
}

CUDATensor::CUDATensor(
    const void* dat, 
    const Shape& s, 
    const lt::dtype dt) :
    sh(s) {
    hipMallocManaged(&arr_, s.elements() * sizeof(data_t));
    hipMemcpy(arr_, dat, s.elements() * sizeof(data_t), hipMemcpyDefault);
    hipDeviceSynchronize();
}
    
std::unique_ptr<TensorGate> CUDATensor::clone() {
    return std::make_unique<CUDATensor>(arr_, sh, lt::dtype::float32);
}

void CUDATensor::assign(const Tensor& t) {
    assert(0 && "not implemented");
}

Tensor CUDATensor::copy() {
    assert(0 && "not implemented");
}

Tensor CUDATensor::shallowCopy() {
    assert(0 && "not implemented");
}

TensorBackend& CUDATensor::backend() const {
    return CUDABackend::getInstance();
}

const Shape& CUDATensor::shape() const {
    return sh;
}

Tensor CUDATensor::index(const Shape& sh) const {
    assert(0 && "not implemented");
}

std::string CUDATensor::toString() const {
    assert(0 && "not implemented");
}

void CUDATensor::buff(void** out) const {
    *out = arr_;
}

data_t* CUDATensor::data() {
    return arr_;
}
const data_t* CUDATensor::data() const {
    return arr_;
}

}  // namespace lt